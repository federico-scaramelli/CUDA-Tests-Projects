#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "../Utils/common.h"

// the kernels prototype
__global__ void blockParReduceUroll(int*, int*, ulong);
__global__ void multBlockParReduceUroll8(int*, int*, ulong);
__global__ void multBlockParReduceUroll16(int*, int*, ulong);


/*
 *  Block by block parallel implementation with divergence
 */
__global__ void blockParReduce1(int* in, int* out, ulong n) {

	uint tid = threadIdx.x;
	ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

	// boundary check
	if (idx >= n)
		return;

	// convert global data pointer to the local pointer of this block
	int* thisBlock = in + blockIdx.x * blockDim.x;

	// in-place reduction in global memory
	for (int stride = 1; stride < blockDim.x; stride *= 2) {
		if ((tid % (2 * stride)) == 0)
			thisBlock[tid] += thisBlock[tid + stride];

		// synchronize within threadblock
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0)
		out[blockIdx.x] = thisBlock[0];
}

/*
 *  Block by block parallel implementation without divergence
 */
__global__ void blockParReduce2(int* in, int* out, ulong n) {

	uint tid = threadIdx.x;
	ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

	// boundary check
	if (idx >= n)
		return;

	// convert global data pointer to the local pointer of this block
	int* thisBlock = in + blockIdx.x * blockDim.x;

	// in-place reduction in global memory
	for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
		if (tid < stride)
			thisBlock[tid] += thisBlock[tid + stride];

		// synchronize within threadblock
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0)
		out[blockIdx.x] = thisBlock[0];
}

/*
 * MAIN: test on parallel reduction
 */
int main(void) {
	int* a, * b, * d_a, * d_b;
	int blockSize = 512;            // block dim 1D
	ulong numBlock = 512 * 512;      // grid dim 1D
	ulong n = blockSize * numBlock;  // array dim
	ulong sum_CPU = 0, sum_GPU;
	ulong size = sizeof(int);
	ulong abc = size;
	const ulong nByte = size * n;
	ulong mByte = numBlock * size;
	double start, stopGPU, stopCPU, speedup;

	printf("\n****  test on parallel reduction  ****\n");

	// init
	a = (int*)malloc(nByte);
	b = (int*)malloc(mByte);
	CHECK(hipMalloc((void**)&d_a, nByte));
	for (ulong i = 0; i < n; i++) a[i] = 1;
	CHECK(hipMemcpy(d_a, a, nByte, hipMemcpyHostToDevice));
	CHECK(hipMalloc((void**)&d_b, mByte));
	CHECK(hipMemset((void*)d_b, 0, mByte));

	/***********************************************************/
	/*                     CPU reduction                       */
	/***********************************************************/
	printf("  Vector length: %.2f MB\n", n / (1024.0 * 1024.0));
	printf("\n  CPU procedure...\n");
	start = seconds();
	for (ulong i = 0; i < n; i++) sum_CPU += a[i];
	stopCPU = seconds() - start;
	printf("    Elapsed time: %f (sec) \n", stopCPU);
	printf("    sum: %lu\n", sum_CPU);

	printf("\n  GPU kernels (mem required %lu bytes)\n", nByte);

	/***********************************************************/
	/*         KERNEL blockParReduce1 (divergent)              */
	/***********************************************************/
	// block by block parallel implementation with divergence
	printf("\n  Launch kernel: blockParReduce1...\n");
	start = seconds();
	blockParReduce1 << <numBlock, blockSize >> > (d_a, d_b, n);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
	stopGPU = seconds() - start;
	speedup = stopCPU / stopGPU;
	printf("    Elapsed time: %f (sec) - speedup %.1f\n", stopGPU, speedup);
	// memcopy D2H
	CHECK(hipMemcpy(b, d_b, mByte, hipMemcpyDeviceToHost));
	// check result
	sum_GPU = 0;
	for (uint i = 0; i < numBlock; i++)
		sum_GPU += b[i];
	assert(sum_GPU == n);
	// reset input vector on GPU
	for (ulong i = 0; i < n; i++) a[i] = 1;
	CHECK(hipMemcpy(d_a, a, nByte, hipMemcpyHostToDevice));

	/***********************************************************/
	/*        KERNEL blockParReduce2  (non divergent)          */
	/***********************************************************/
	// block by block parallel implementation without divergence
	printf("\n  Launch kernel: blockParReduce2...\n");
	start = seconds();
	blockParReduce2 << <numBlock, blockSize >> > (d_a, d_b, n);
	CHECK(hipDeviceSynchronize());
	stopGPU = seconds() - start;
	speedup = stopCPU / stopGPU;
	printf("    Elapsed time: %f (sec) - speedup %.1f\n", stopGPU, speedup);
	CHECK(hipGetLastError());
	// memcopy D2H
	CHECK(hipMemcpy(b, d_b, mByte, hipMemcpyDeviceToHost));
	// check result
	sum_GPU = 0;
	for (uint i = 0; i < numBlock; i++) {
		sum_GPU += b[i];
		//		printf("b[%d] = %d\n",i,b[i]);
	}
	assert(sum_GPU == n);
	// reset input vector on GPU
	for (ulong i = 0; i < n; i++) a[i] = 1;
	CHECK(hipMemcpy(d_a, a, nByte, hipMemcpyHostToDevice));

	/***********************************************************/
	/*               KERNEL blockParReduceUroll                */
	/***********************************************************/
	// block by block parallel implementation without divergence
	printf("\n  Launch kernel: blockParReduceUroll...\n");
	start = seconds();
	blockParReduceUroll << <numBlock, blockSize >> > (d_a, d_b, n);
	CHECK(hipDeviceSynchronize());
	stopGPU = seconds() - start;
	speedup = stopCPU / stopGPU;
	printf("    Elapsed time: %f (sec) - speedup %.1f\n", stopGPU, speedup);
	CHECK(hipGetLastError());
	// memcopy D2H
	CHECK(hipMemcpy(b, d_b, mByte, hipMemcpyDeviceToHost));
	// check result
	sum_GPU = 0;
	for (uint i = 0; i < numBlock; i++)
		sum_GPU += b[i];
	assert(sum_GPU == n);
	// reset input vector on GPU
	for (ulong i = 0; i < n; i++) a[i] = 1;
	CHECK(hipMemcpy(d_a, a, nByte, hipMemcpyHostToDevice));

	/***********************************************************/
	/*            KERNEL multBlockParReduceUroll8              */
	/***********************************************************/
	// block by block parallel implementation without divergence
	printf("\n  Launch kernel: multBlockParReduceUroll8...\n");
	start = seconds();
	multBlockParReduceUroll8 << <numBlock / 8, blockSize >> > (d_a, d_b, n);
	CHECK(hipDeviceSynchronize());
	stopGPU = seconds() - start;
	speedup = stopCPU / stopGPU;
	printf("    Elapsed time: %f (sec) - speedup %.1f\n", stopGPU, speedup);
	CHECK(hipGetLastError());
	// memcopy D2H
	CHECK(hipMemcpy(b, d_b, mByte, hipMemcpyDeviceToHost));
	// check result
	sum_GPU = 0;
	for (uint i = 0; i < numBlock / 8; i++)
		sum_GPU += b[i];
	printf("    sum: %lu\n", sum_GPU);
	assert(sum_GPU == n);
	// reset input vector on GPU
	for (ulong i = 0; i < n; i++) a[i] = 1;
	CHECK(hipMemcpy(d_a, a, nByte, hipMemcpyHostToDevice));

	/***********************************************************/
	/*            KERNEL multBlockParReduceUroll16             */
	/***********************************************************/
	// block by block parallel implementation without divergence
	printf("\n  Launch kernel: multBlockParReduceUroll16...\n");
	start = seconds();
	multBlockParReduceUroll16 << <numBlock / 16, blockSize >> > (d_a, d_b, n);
	CHECK(hipDeviceSynchronize());
	stopGPU = seconds() - start;
	speedup = stopCPU / stopGPU;
	printf("    Elapsed time: %f (sec) - speedup %.1f\n", stopGPU, speedup);
	CHECK(hipGetLastError());
	// memcopy D2H
	CHECK(hipMemcpy(b, d_b, mByte, hipMemcpyDeviceToHost));
	// check result
	sum_GPU = 0;
	for (uint i = 0; i < numBlock / 16; i++)
		sum_GPU += b[i];
	assert(sum_GPU == n);

	hipFree(d_a);

	CHECK(hipDeviceReset());
	return 0;
}

