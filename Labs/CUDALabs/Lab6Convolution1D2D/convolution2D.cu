#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "../Utils/common.h"

#define DATA_WIDTH   (20*1024)
#define DATA_HEIGHT  (20*1024)
#define BLOCK_SIZE   8
#define MASK_RADIUS  2
#define MASK_SIZE    (2 * MASK_RADIUS + 1)
#define TILE_WIDTH   (BLOCK_SIZE + MASK_SIZE - 1)
#define DEBUG 0

// constant mem
__constant__ float M_dev[MASK_SIZE * MASK_SIZE];

/*
 * kernel for convolution 2D (it holds only if MASK_RADIUS < BLOCK_SIZE)
 */
__global__ void conv2D(float* A, float* B) {

	if (MASK_RADIUS >= BLOCK_SIZE)
		return;

	const int TILE_SIZE = TILE_WIDTH + MASK_RADIUS * 2;

	__shared__ float A_s[TILE_SIZE][TILE_SIZE];

	//edge left ALONE
	if (threadIdx.x < MASK_RADIUS) {
		A_s[threadIdx.y + MASK_RADIUS][threadIdx.x] = A[threadIdx.y * blockDim.x + threadIdx.x];
	}
	//edge right ALONE
	if (threadIdx.x >= BLOCK_SIZE - MASK_RADIUS) {
		A_s[threadIdx.y + MASK_RADIUS][threadIdx.x + MASK_RADIUS * 2] = A[threadIdx.y * blockDim.x + threadIdx.x];
	}
	//edge top ALONE
	if (threadIdx.y < MASK_RADIUS) {
		A_s[threadIdx.y][threadIdx.x + MASK_RADIUS] = A[threadIdx.y * blockDim.x + threadIdx.x];
	}
	//edge bottom ALONE
	if (threadIdx.y >= BLOCK_SIZE - MASK_RADIUS) {
		A_s[threadIdx.y + MASK_RADIUS * 2][threadIdx.x + MASK_RADIUS] = A[threadIdx.y * blockDim.x + threadIdx.x];
	}

	//Corner top left
	if (threadIdx.x < MASK_RADIUS && threadIdx.y < MASK_RADIUS) {
		A_s[threadIdx.y][threadIdx.x] = A[threadIdx.y * blockDim.x + threadIdx.x];
	}
	//Corner top right
	if (threadIdx.y < MASK_RADIUS && threadIdx.x >= MASK_RADIUS) {
		A_s[threadIdx.y][threadIdx.x + MASK_RADIUS * 2] = A[threadIdx.y * blockDim.x + threadIdx.x];
	}
	//Corner bottom left
	if (threadIdx.y >= BLOCK_SIZE - MASK_RADIUS && threadIdx.x < MASK_RADIUS) {
		A_s[threadIdx.y + MASK_RADIUS * 2][threadIdx.x] = A[threadIdx.y * blockDim.x + threadIdx.x];
	}
	//Corner bottom right
	if (threadIdx.y >= BLOCK_SIZE - MASK_RADIUS && threadIdx.x >= BLOCK_SIZE - MASK_RADIUS) {
		A_s[threadIdx.y + MASK_RADIUS * 2][threadIdx.x + MASK_RADIUS * 2] = A[threadIdx.y * blockDim.x + threadIdx.x];
	}

	//Center (all the matrix 1:1)
	A_s[threadIdx.y + MASK_RADIUS][threadIdx.x + MASK_RADIUS] = A[threadIdx.y * blockDim.x + threadIdx.x];

	__syncthreads();

	float out_val = 0;
	for (int i = 0; i < MASK_SIZE; i++) {
		for (int j = 0; j < MASK_SIZE; j++) {
			out_val += A_s[threadIdx.y + i][threadIdx.x + j] * M_dev[j * MASK_SIZE + i];
		}
	}
	
	B[threadIdx.y * blockDim.x + threadIdx.x] = out_val;
}

/*
 * Average filter
 */
void Avg_mask(float* mask) {
	int n = MASK_SIZE;
	for (int i = 0; i < n * n; i++)
		mask[i] = (float)1.0 / (n * n);
}


/*
 * main
 */
int main(void) {

	// check params
	if (MASK_RADIUS >= BLOCK_SIZE) {
		printf("ERROR: it holds only if MASK_RADIUS < BLOCK_SIZE!\n");
		return 1;
	}

	int nW = DATA_WIDTH;
	int nH = DATA_HEIGHT;
	int b = BLOCK_SIZE;

	float M[MASK_SIZE * MASK_SIZE]; // const size
	float* A, * B, * A_dev, * B_dev;
	int datasize = nW * nH * sizeof(float);
	int masksize = MASK_SIZE * MASK_SIZE * sizeof(float);

	printf("Data size: %.2f (MB)\n", (float)datasize / (1024.0 * 1024.0));
	printf("Initializing data...\n");
	A = (float*)malloc(datasize);
	B = (float*)malloc(datasize);

	// initialize data
	for (int i = 0; i < nH; i++)
		for (int j = 0; j < nW; j++)
			A[i * nW + j] = rand() % 10;

	// initialize mask 
	Avg_mask(M);

#if DEBUG
	// print data
	printf("Print matrix A...\n");
	for (int i = 0; i < nH; i++) {
		if (i % 8 == 0 && i > 0)
			printf("\n");

		for (int j = 0; j < nW; j++)
			if (j % 8 == 0 && j > 0)
				printf(" %0.0f ", A[i * nW + j]);
			else
				printf("%0.0f ", A[i * nW + j]);
		printf("\n");
	}

	printf("Print matrix M ...\n");
	for (int i = 0; i < MASK_SIZE; i++) {
		for (int j = 0; j < MASK_SIZE; j++)
			printf(" %1.2f ", M[i * MASK_SIZE + j]);
		printf("\n");
	}
#endif

	// cuda allocation 
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(M_dev), M, masksize));
	CHECK(hipMalloc((void**)&A_dev, datasize));
	CHECK(hipMalloc((void**)&B_dev, datasize));
	CHECK(hipMemcpy(A_dev, A, datasize, hipMemcpyHostToDevice));

	// block, grid dims, kernel
	dim3 block(b, b);
	dim3 grid((nW + b - 1) / b, (nH + b - 1) / b);
	double iStart, iElaps;
	iStart = seconds();
	conv2D << <grid, block >> > (A_dev, B_dev);
	hipDeviceSynchronize();
	iElaps = seconds() - iStart;
	printf("\nconv2D<<<(%d,%d), (%d,%d)>>> elapsed time %f sec \n\n", grid.x, grid.y, block.x, block.y, iElaps);
	CHECK(hipGetLastError());

	CHECK(hipMemcpy(B, B_dev, datasize, hipMemcpyDeviceToHost));

#if DEBUG
	// print out data
	printf("Print results...\n");
	for (int i = 0; i < nH; i++) {
		if (i % 8 == 0 && i > 0)
			printf("\n");
		for (int j = 0; j < nW; j++)
			if (j % 8 == 0 && j > 0)
				printf(" %0.2f ", B[i * nW + j]);
			else
				printf("%0.2f ", B[i * nW + j]);
		printf("\n");
	}
#endif

	hipFree(A_dev);
	hipFree(B_dev);
	hipDeviceReset();
	free(A);
	free(B);
	return 0;
}