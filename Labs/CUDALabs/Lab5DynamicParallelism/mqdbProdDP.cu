#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "../Utils/mqdb/mqdb.h"

#define BLOCK_SIZE 16     // block size


/*
 * Kernel for block sub-matrix product of mqdb
 */
__global__ void mqdbBlockProd(mqdb A, mqdb B, mqdb C, uint sdim, uint d, uint n) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// jump to the right block sub-matrix
	int  offset = (n + 1) * sdim;

	// each thread computes an entry of the product matrix
	if ((row < d) && (col < d)) {
		float val = 0;

		for (int k = 0; k < d; k++)
			val += A.elem[row * n + k + offset] * B.elem[k * n + col + offset];
		C.elem[row * n + col + offset] = val;
	}
}

/*
 * Kernel for block sub-matrix product of mqdb: parent grid(1)
 */
__global__ void mqdbProdDP1(mqdb A, mqdb B, mqdb C, uint k, uint n) {

	// TODO
}

/*
 * Kernel for block sub-matrix product of mqdb: parent grid(k)
 */
__global__ void mqdbProdDPk(mqdb A, mqdb B, mqdb C, uint n) {

	// TODO

}