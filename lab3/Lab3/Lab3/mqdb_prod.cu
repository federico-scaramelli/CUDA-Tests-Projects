#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "common.h"
#include "mqdb.h"

#define BLOCK_SIZE 16     // block size

struct tms {
	double CPUtms;
	double GPUtmsNaive;
	double GPUtmsMQDB;
	float density;
};


__global__ void matProd(mqdb A, mqdb B, mqdb C, int n) {
	// row & col indexes
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// each thread computes an entry of the product matrix
	if ((row < n) && (col < n)) {
		float val = 0;
		for (int k = 0; k < n; k++)
			val += A.elem[row * n + k] * B.elem[k * n + col];
		C.elem[row * n + col] = val;
	}
}


__global__ void mqdbBlockProd(mqdb A, mqdb B, mqdb C, int n, uint blockNumber) {

	// TODO
	uint row = blockIdx.y * blockDim.y + threadIdx.y;
	uint col = blockIdx.x * blockDim.x + threadIdx.x;

	//printf("Block Idx.x: %d; Block Idx.y: %d; \n", blockIdx.x, blockIdx.y);

	int startBlockIndex = 0;
	for (int i = 0; i < blockNumber; i++) {
		startBlockIndex += A.blkSize[i] * n + A.blkSize[i];
	}

	//printf("Start index: %d\n",startBlockIndex);

	// each thread computes an entry of the product matrix
	if ((row < A.blkSize[blockNumber]) && (col < A.blkSize[blockNumber])) {
		float val = 0;
		for (int k = 0; k < A.blkSize[blockNumber]; k++) {
			val += A.elem[startBlockIndex + row * n + k] * B.elem[startBlockIndex + k * n + col];
			// printf("MatA[%d] * MatB[%d] = %5.2f\n", startBlockIndex + row * n + k, startBlockIndex + k * n + col, A.elem[startBlockIndex + row * n + k] * B.elem[startBlockIndex + k * n + col]);
		}
		// printf("AGGIORNO MatC[%d] = %5.2f\n", startBlockIndex + row * n + col, val);
		C.elem[startBlockIndex + row * n + col] = val;
	}
}


void testKernelsMQDB(uint n, uint k, struct tms* times) {

	// mqdb host matrices
	mqdb A, B, C, C1, C2;

	// mqdb device matrices
	mqdb d_A, d_B, d_C;

	// fill in
	A = mqdbConst(n, k, 10, 1);
	B = mqdbConst(n, k, 10, 1);
	C = mqdbConst(n, k, 10, 1);
	C1 = mqdbConst(n, k, 10, 1);
	C2 = mqdbConst(n, k, 10, 1);

	ulong nBytes = n * n * sizeof(float);
	ulong kBytes = k * sizeof(uint);
	printf("Memory size required = %.1f (MB)\n", static_cast<float>(nBytes) / (1024.0 * 1024.0));

	// malloc and copy on device memory
	d_A.nBlocks = A.nBlocks;
	CHECK(hipMalloc((void**)&d_A.blkSize, kBytes));
	CHECK(hipMemcpy(d_A.blkSize, A.blkSize, kBytes, hipMemcpyHostToDevice));
	CHECK(hipMalloc((void**)&d_A.elem, nBytes));
	CHECK(hipMemcpy(d_A.elem, A.elem, nBytes, hipMemcpyHostToDevice));
	d_B.nBlocks = B.nBlocks;
	CHECK(hipMalloc((void**)&d_B.blkSize, kBytes));
	CHECK(hipMemcpy(d_B.blkSize, B.blkSize, kBytes, hipMemcpyHostToDevice));
	CHECK(hipMalloc((void**)&d_B.elem, nBytes));
	CHECK(hipMemcpy(d_B.elem, B.elem, nBytes, hipMemcpyHostToDevice));
	d_C.nBlocks = C.nBlocks;
	CHECK(hipMalloc((void**)&d_C.blkSize, kBytes));
	CHECK(hipMemcpy(d_C.blkSize, C.blkSize, kBytes, hipMemcpyHostToDevice));
	CHECK(hipMalloc((void**)&d_C.elem, nBytes));
	CHECK(hipMemset(d_C.elem, 0.0, nBytes));


	//                    CPU MQDB product
	printf("CPU MQDB product...\n");
	double start = seconds();
	mqdbProd(A, B, C);
	double CPUTime = seconds() - start;
	printf("   CPU elapsed time: %.5f (sec)\n\n", CPUTime);


	//                     GPU mat product 
	printf("Kernel (naive) mat product...\n");
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((n + block.x - 1) / block.x, (n + block.y - 1) / block.y);
	start = seconds();
	matProd <<<grid, block >>> (d_A, d_B, d_C, n);
	CHECK(hipDeviceSynchronize());
	double GPUtime1 = seconds() - start;
	printf("   elapsed time:                %.2f (sec)\n", GPUtime1);
	printf("   speedup vs CPU MQDB product: %.2f\n", CPUTime / GPUtime1);
	CHECK(hipMemcpy(C1.elem, d_C.elem, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipMemset(d_C.elem, 0.0, nBytes));
	checkResult(C, C1);
	//	mqdbDisplay(C1);
	
	//                     GPU MQDB product
	printf("Kernel MQDB product...\n");

	// TODO
	start = seconds();

	for (uint i = 0; i < A.nBlocks; i++) {
		grid.x = (A.blkSize[i] + block.x - 1) / block.x;
		grid.y = (A.blkSize[i] + block.y - 1) / block.y;
		mqdbBlockProd <<<grid, block >>> (d_A, d_B, d_C, n, i);
	}
	CHECK(hipDeviceSynchronize());
	double GPUtime2 = seconds() - start;
	printf("   elapsed time:                    %.2f (sec)\n", GPUtime2);
	printf("   speedup vs CPU MQDB product:     %.2f\n", CPUTime / GPUtime2);
	printf("   speedup vs GPU std mat product:  %.2f\n", GPUtime1 / GPUtime2);
	// copy the array 'C' back from the GPU to the CPU
	CHECK(hipMemcpy(C2.elem, d_C.elem, nBytes, hipMemcpyDeviceToHost));
	CHECK(hipMemset(d_C.elem, 0.0, nBytes));
	// mqdbDisplay(&C);
	// mqdbDisplay(&C2);
	checkResult(C, C2);

	CHECK(hipFree(d_A.elem));
	CHECK(hipFree(d_B.elem));
	CHECK(hipFree(d_C.elem));

	// collect times
	times->CPUtms = CPUTime;
	times->GPUtmsNaive = GPUtime1;
	times->GPUtmsMQDB = GPUtime2;

	float den = 0;
	for (uint j = 0; j < k; j++)
		den += A.blkSize[j] * A.blkSize[j];
	times->density = den / (n * n);
}

int main(int argc, char* argv[]) {
	uint n = 2*1024;      // matrix size
	const uint min_k = 30;       // max num of blocks
	const uint max_k = 30;       // max num of blocks
	// uint n = 30;      // matrix size
	// const uint min_k = 2;       // max num of blocks
	// const uint max_k = 2;       // max num of blocks

	struct tms times[max_k - min_k + 1];

	// multiple tests on kernels
	for (uint k = min_k; k <= max_k; k++) {
		printf("\n*****   k = %d --- (avg block size = %f)\n", k, static_cast<float>(n) / k);
		testKernelsMQDB(n, k, &times[k - min_k]);
	}

	FILE* fd;
	fd = fopen("res.csv", "w");
	if (fd == NULL) {
		perror("file error!\n");
		exit(1);
	}

	// write results on file
	fprintf(fd, "num blocks,");
	for (uint j = 0; j <= max_k - min_k; j++)
		fprintf(fd, "%d,", j + min_k);

	fprintf(fd, "\nCPU MQDB product,");
	for (uint j = 0; j <= max_k - min_k; j++)
		fprintf(fd, "%.4f,", times[j].CPUtms);

	fprintf(fd, "\nKernel mat product naive,");
	for (uint j = 0; j <= max_k - min_k; j++)
		fprintf(fd, "%.4f,", times[j].GPUtmsNaive);

	fprintf(fd, "\nKernel MQDB product,");
	for (uint j = 0; j <= max_k - min_k; j++)
		fprintf(fd, "%.4f,", times[j].GPUtmsMQDB);

	fprintf(fd, "\ndensity,");
	for (uint j = 0; j <= max_k - min_k; j++)
		fprintf(fd, "%.4f,", times[j].density);

	fclose(fd);

	return 0;
}
